#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>

extern __device__ bool __heq(__half arg1, __half arg2);

__device__
__half test_add_fp16(__half arg1, __half arg2)
{
  return __hadd(arg1, arg2);
}

__device__
bool test_cmp_fp16(__half arg1, __half arg2)
{
  return __heq(arg1, arg2);
}
